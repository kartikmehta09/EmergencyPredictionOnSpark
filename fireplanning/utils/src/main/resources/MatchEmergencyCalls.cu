
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>


#define RADIUS_IN_KM 6372.8


extern "C"
// Computes the haversine distance betwwen two points on Earth
__global__ void haversine(int *size, double *in, double *out) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < *size ) {
	    const int lat1ix = 4*ix,lon1ix = (4*ix)+1,lat2ix = (4*ix)+2, lon2ix = (4*ix)+3;
	    const double dLat = (in[lat2ix] - in[lat1ix] ) * (HIP_PI_F /180.0);
	    const double dLon = (in[lon2ix] - in[lon1ix] ) * (HIP_PI_F /180.0);
	    const double a = pow(sin(dLat/2.0),2.0) + pow(sin(dLon/2.0),2.0) * cos(in[lat1ix] * (HIP_PI_F/180.0)) * cos(in[lat2ix] * (HIP_PI_F/180.0));
	    const double c = 2.0 * asin(sqrt(a));
    	    out[ix] = RADIUS_IN_KM * c;
    }
}

